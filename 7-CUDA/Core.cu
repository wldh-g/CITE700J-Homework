#include "Core_CUDA.cuh"

void __exec_base(std::function<void(void)> c1_func, std::function<void(void)> c2_func,
                 std::function<void(void)> cuda_func, std::function<void(void)> c1_flush,
                 std::function<void(void)> c2_flush, std::function<void(void)> cuda_flush,
                 std::function<void(double, const char*)> c1_report,
                 std::function<void(double, const char*)> c2_report,
                 std::function<void(double, const char*)> cuda_report,
                 bool c1_enable, bool c2_enable, bool cuda_enable, size_t loop_max) {
  CPerfCounter timer;
  double c1_time = 0;
  double c2_time = 0;
  double cuda_time = 0;

  if (c1_enable) {
    try {
      for (size_t loop_cnt = 0; loop_cnt < loop_max; loop_cnt += 1) {
        timer.Reset();
        c1_flush();
        timer.Start();
        c1_func();
        timer.Stop();
        c1_time += timer.GetElapsedTime();
      }
      c1_report(c1_time / (double)loop_max * 1000.0, nullptr);
    } catch (const char* error) {
      c1_report(0, error);
    } catch (...) {
      c1_report(0, "Unknown error occurred");
    }
  }

  if (c2_enable) {
    try {
      for (size_t loop_cnt = 0; loop_cnt < loop_max; loop_cnt += 1) {
        timer.Reset();
        c2_flush();
        timer.Start();
        c2_func();
        timer.Stop();
        c2_time += timer.GetElapsedTime();
      }
      c2_report(c2_time / (double)loop_max * 1000.0, nullptr);
    } catch (const char* error) {
      c2_report(0, error);
    } catch (...) {
      c2_report(0, "Unknown error occurred");
    }
  }

  if (cuda_enable) {
    try {
      // Set up device
      int cuda_device_count = 0;
      if (hipGetDeviceCount(&cuda_device_count) != hipSuccess || cuda_device_count < 1)
        throw "Failed to get CUDA device";
      if (hipSetDevice(0) != hipSuccess) throw "Failed to set CUDA device to 0";

      // Do execution
      for (size_t loop_cnt = 0; loop_cnt < loop_max; loop_cnt += 1) {
        timer.Reset();
        timer.Start();
        cuda_func();
        hipDeviceSynchronize();
        timer.Stop();
        cuda_time += timer.GetElapsedTime();
        cuda_flush();
      }

      // Clearing out
      if (hipDeviceReset() != hipSuccess) throw "Failed to reset CUDA device";
      cuda_report(cuda_time / (double)loop_max * 1000.0, nullptr);
    } catch (const char* error) {
      cuda_flush();
      cuda_report(0, error);
    } catch (...) {
      cuda_flush();
      cuda_report(0, "Unknown error occurred");
    }
  }
};
