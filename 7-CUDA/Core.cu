#include "hip/hip_runtime.h"
#include "Core_CUDA.cuh"

void __exec_base(std::function<void(CPerfCounter&)> c1_func,
                 std::function<void(CPerfCounter&)> c2_func,
                 std::function<void(CPerfCounter&)> cuda_func, std::function<void(void)> c1_flush,
                 std::function<void(void)> c2_flush, std::function<void(void)> cuda_flush,
                 std::function<void(double, const char*)> c1_report,
                 std::function<void(double, const char*)> c2_report,
                 std::function<void(double, const char*)> cuda_report,
                 bool c1_enable, bool c2_enable, bool cuda_enable, size_t loop_max) {
  CPerfCounter timer;
  double c1_time = 0;
  double c2_time = 0;
  double cuda_time = 0;

  if (c1_enable) {
    try {
      for (size_t loop_cnt = 0; loop_cnt < loop_max; loop_cnt += 1) {
        timer.Reset();
        c1_flush();
        c1_func(timer);
        c1_time += timer.GetElapsedTime();
      }
      c1_report(c1_time / (double)loop_max * 1000.0, nullptr);
    } catch (const char* error) {
      c1_report(0, error);
    } catch (...) {
      c1_report(0, "Unknown error occurred");
    }
  } else { c1_report(std::numeric_limits<double>::max(), nullptr); }

  if (c2_enable) {
    try {
      for (size_t loop_cnt = 0; loop_cnt < loop_max; loop_cnt += 1) {
        timer.Reset();
        c2_flush();
        c2_func(timer);
        c2_time += timer.GetElapsedTime();
      }
      c2_report(c2_time / (double)loop_max * 1000.0, nullptr);
    } catch (const char* error) {
      c2_report(0, error);
    } catch (...) {
      c2_report(0, "Unknown error occurred");
    }
  } else { c2_report(std::numeric_limits<double>::max(), nullptr); }

  if (cuda_enable) {
    try {
      // Set up device
      int cuda_device_count = 0;
      if (hipGetDeviceCount(&cuda_device_count) != hipSuccess || cuda_device_count < 1)
        throw "Failed to get CUDA device";
      if (hipSetDevice(0) != hipSuccess) throw "Failed to set CUDA device to 0";

      // Do execution
      for (size_t loop_cnt = 0; loop_cnt < loop_max; loop_cnt += 1) {
        timer.Reset();
        cuda_func(timer);
        cuda_time += timer.GetElapsedTime();
        cuda_flush();
      }

      // Clearing out
      if (hipDeviceReset() != hipSuccess) throw "Failed to reset CUDA device";
      cuda_report(cuda_time / (double)loop_max * 1000.0, nullptr);
    } catch (hipError_t& error_code) {
      cuda_report(0, ("Failed to synchronize CUDA core after launching the kernel ("
                      + std::to_string(error_code) + ")").c_str());
    } catch (const char* error) {
      cuda_flush();
      cuda_report(0, error);
    } catch (...) {
      cuda_flush();
      cuda_report(0, "Unknown error occurred");
    }
  } else { cuda_report(std::numeric_limits<double>::max(), nullptr); }
};

__global__ void setKernel_int8_global(filt::Filter<int8_t>* filter, int8_t* kernel_ptr) {
  filter->kernel = kernel_ptr;
};
void setKernel(filt::Filter<int8_t>* filter, int8_t* kernel_ptr) {
  setKernel_int8_global __set_block_thread__(1,1) (filter, kernel_ptr);
};
