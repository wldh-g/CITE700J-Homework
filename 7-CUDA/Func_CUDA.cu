#include "hip/hip_runtime.h"
#include "Core.h"
#include "Functions.cuh"
#include "Util.cuh"

/********************
 ** CUDA Functions **
 ********************/

///////////////
// Inversion //
///////////////

__global__ void invert_8b(uint8_t* in, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = y_pos * x_size + x_pos;
  
  if (x_pos < x_size) {
    out[index] = 255 - in[index];
  }
};
void cuda::invert_8b __make_host__(invert_8b, uint8_t, uint8_t);

//////////////////
// Accumulation //
//////////////////

void cuda::accumulation_16b __cuda_todo__(accumulation_16b, uint16_t, uint64_t);

//////////////
// Flipping //
//////////////

__global__ void xflip(uint8_t* in, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;

  if (x_pos < x_size) {
    out[y_pos * x_size + x_size - x_pos - 1] = in[x_size * y_pos + x_pos];
  }
};
void cuda::xflip __make_host__(xflip, uint8_t, uint8_t);

__global__ void yflip(uint8_t* in, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;

  if (x_pos < x_size) {
    out[(y_size - y_pos - 1) * x_size + x_pos] = in[x_size * y_pos + x_pos];
  }
};
void cuda::yflip __make_host__(yflip, uint8_t, uint8_t);

//////////////
// Addition //
//////////////

__global__ void add_8b(uint8_t* in1, uint8_t* in2, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;

  if (x_pos < x_size) {
    out[index] = in1[index] / 2 + in2[index] / 2;
  }
};
void cuda::add_8b __make_host_2__(add_8b, uint8_t, uint8_t);

__global__ void add_16b(uint8_t* in1, uint8_t* in2, uint16_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;

  if (x_pos < x_size) {
    out[index] = (uint16_t)in1[index] + (uint16_t)in2[index];
  }
};
void cuda::add_16b __make_host_2__(add_16b, uint8_t, uint16_t);

/////////////////
// Dot Product //
/////////////////

__global__ void dot(uint8_t* in1, uint8_t* in2, uint64_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;
  __block__ uint64_t bob;
  bob = 0;
  __syncthreads();

  if (x_pos < x_size) {
    atomicAdd(&bob, (uint64_t)in1[index] * (uint64_t)in2[index]);
  }
  __syncthreads();

  __for_each_block__ {
    atomicAdd(out, bob);
  }
};
void cuda::dot __make_host_2__(dot, uint8_t, uint64_t);

///////////////
// Histogram //
///////////////

__global__ void histogram_8bin(uint16_t* in, uint64_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  __block__ uint64_t carol[8];
  carol[x_pos % 8] = 0;
  __syncthreads();
  atomicAdd(carol + (in[x_pos + __y_pos__ * x_size] % 8), 1);
  __syncthreads();
  if (threadIdx.x == 0) {
    atomicAdd(out, *(carol));
    atomicAdd(out + 1, *(carol + 1));
    atomicAdd(out + 2, *(carol + 2));
    atomicAdd(out + 3, *(carol + 3));
    atomicAdd(out + 4, *(carol + 4));
    atomicAdd(out + 5, *(carol + 5));
    atomicAdd(out + 6, *(carol + 6));
    atomicAdd(out + 7, *(carol + 7));
  }
};
void cuda::histogram_8bin __make_host__(histogram_8bin, uint16_t, uint64_t);

///////////////
// Transpose //
///////////////

__global__ void transpose8(uint8_t* in, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;

  if (x_pos < x_size) {
    out[x_pos * x_size + y_pos] = in[x_size * y_pos + x_pos];
  }
};
void cuda::transpose_line_by_line __make_host__(transpose8, uint8_t, uint8_t);
void cuda::transpose_block_8 __make_host__(transpose8, uint8_t, uint8_t);
void cuda::transpose_block_16 __make_host__(transpose8, uint8_t, uint8_t);
void cuda::transpose_block_32 __make_host__(transpose8, uint8_t, uint8_t);
void cuda::transpose_block_64 __make_host__(transpose8, uint8_t, uint8_t);
void cuda::transpose_block_128 __make_host__(transpose8, uint8_t, uint8_t);

__global__ void transpose16(uint16_t* in, uint16_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;

  if (x_pos < x_size) {
    out[x_pos * x_size + y_pos] = in[x_size * y_pos + x_pos];
  }
};
void cuda::transpose16_block_8 __make_host__(transpose16, uint16_t, uint16_t);

__global__ void transpose32(uint32_t* in, uint32_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;

  if (x_pos < x_size) {
    out[x_pos * x_size + y_pos] = in[x_size * y_pos + x_pos];
  }
};
void cuda::transpose32_block_4 __make_host__(transpose32, uint32_t, uint32_t);

/////////////////
// Convolution //
/////////////////

__global__ void conv_zp_unsigned(uint8_t* in, const filt::Filter<int8_t>* filter, uint8_t* out,
                                 size_t x_size, size_t y_size) {
  /* 
   * This convolution implements element-wise convolution and uses exact number of threads. [1 & 2]
   * Below assumptions are considered in this convolution implementation.
   *  - Assumption 1 : GPU cannot expect the size and traits of the filter.
   *  - Assumption 2 : The only known fact is that the shape of the filter is square.
   *  - Assumption 3 : GPU cannot expect the size and traits of the image.
   */
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;
  uint32_t x_bud = blockDim.x + 2 * filter->bud_size;
  extern __block__ uint8_t alice[]; // shape : (blockDim.x + filter->size - 1, filter->size)
  
  // Set all zero to all
  for (size_t i = 0; i < filter->size2; i += 1) {
    alice[threadIdx.x * filter->size + i] = 0;
  }
  __syncthreads();

  // Fill the alice
  size_t alice_x_size = blockDim.x + filter->size - 1;
  for (int alice_y_pos = 0; alice_y_pos < filter->size; alice_y_pos += 1) {
    int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + alice_y_pos;
    if (src_y_pos >= 0 && src_y_pos < y_size) {
      alice[alice_y_pos * alice_x_size + threadIdx.x + filter->bud_size]
        = in[src_y_pos * x_size + x_pos];
    }
  }
  if (blockIdx.x != 0 && threadIdx.x < filter->bud_size) {
    for (int alice_y_pos = 0; alice_y_pos < filter->size; alice_y_pos += 1) {
      int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + alice_y_pos;
      if (src_y_pos >= 0 && src_y_pos < y_size) {
        alice[alice_y_pos * alice_x_size + threadIdx.x] = in[src_y_pos * x_size +
          (size_t)blockDim.x * blockIdx.x - filter->bud_size + threadIdx.x];
      }
    }
  }
  if ((gridDim.x - 1) > blockIdx.x && threadIdx.x < filter->bud_size) {
    for (int alice_y_pos = 0; alice_y_pos < filter->size; alice_y_pos += 1) {
      int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + alice_y_pos;
      if (src_y_pos >= 0 && src_y_pos < y_size) {
        alice[((size_t)alice_y_pos + 1) * alice_x_size - filter->bud_size + threadIdx.x]
          = in[src_y_pos * x_size + ((size_t)blockIdx.x + 1) * blockDim.x + threadIdx.x];
      }
    }
  }
  __syncthreads();

  // Calculate convolution
  int32_t sum = 0;
  for (size_t fy = 0; fy < filter->size; fy += 1) {
    for (size_t fx = 0; fx < filter->size; fx += 1) {
      sum += filter->kernel[fy * filter->size + fx] * alice[threadIdx.x + fy * x_bud + fx];
    }
  }
  sum /= filter->kernel_sum;
  *(out + index) = sum >= 0 ? (sum <= 255 ? sum : 255) : 0;
};
void cuda::conv_zp_unsigned __make_host_f__(conv_zp_unsigned, uint8_t, int8_t, uint8_t);

__global__ void conv_zb_unsigned(uint8_t* in, const filt::Filter<int8_t>* filter, uint8_t* out,
                                 size_t x_size, size_t y_size) {
  /*
   * This convolution implements element-wise convolution using bigger threads block. [3]
   * Below assumptions are considered in this convolution implementation.
   *  - Assumption 1 : GPU cannot expect the size and traits of the filter.
   *  - Assumption 2 : The only known fact is that the shape of the filter is square.
   *  - Assumption 3 : GPU cannot expect the size and traits of the image.
   */
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;
  uint32_t x_bud = blockDim.x + 2 * filter->bud_size;
  int alice_y_pos = threadIdx.z;
  extern __block__ uint8_t alice[]; // shape : (blockDim.x + filter->size - 1, filter->size)

  // Set all zero to all
  for (size_t i = 0; i < filter->size2; i += 1) {
    alice[threadIdx.x * filter->size + i] = 0;
  }
  __syncthreads();

  // Fill the alice
  size_t alice_x_size = blockDim.x + filter->size - 1;
  int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + alice_y_pos;
  if (src_y_pos >= 0 && src_y_pos < y_size) {
    alice[alice_y_pos * alice_x_size + threadIdx.x + filter->bud_size]
      = in[src_y_pos * x_size + x_pos];
  }
  if (blockIdx.x != 0 && threadIdx.x < filter->bud_size) {
    int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + alice_y_pos;
    if (src_y_pos >= 0 && src_y_pos < y_size) {
      alice[alice_y_pos * alice_x_size + threadIdx.x] = in[src_y_pos * x_size +
        (size_t)blockDim.x * blockIdx.x - filter->bud_size + threadIdx.x];
    }
  }
  if ((gridDim.x - 1) > blockIdx.x && threadIdx.x < filter->bud_size) {
    int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + alice_y_pos;
    if (src_y_pos >= 0 && src_y_pos < y_size) {
      alice[((size_t)alice_y_pos + 1) * alice_x_size - filter->bud_size + threadIdx.x]
        = in[src_y_pos * x_size + ((size_t)blockIdx.x + 1) * blockDim.x + threadIdx.x];
    }
  }
  __syncthreads();

  // Calculate convolution
  int32_t sum = 0;
  for (size_t fy = 0; fy < filter->size; fy += 1) {
    for (size_t fx = 0; fx < filter->size; fx += 1) {
      sum += filter->kernel[fy * filter->size + fx] * alice[threadIdx.x + fy * x_bud + fx];
    }
  }
  sum /= filter->kernel_sum;
  *(out + index) = sum >= 0 ? (sum <= 255 ? sum : 255) : 0;
};
void cuda::conv_zb_unsigned __make_host_f__(conv_zb_unsigned, uint8_t, int8_t, uint8_t);

void cuda::conv_zp_signed __cuda_todo_f__(conv_zp_signed, uint8_t, int8_t, int8_t);

__global__ void conv_be_unsigned(uint8_t* in, const filt::Filter<int8_t>* filter, uint8_t* out,
                                 size_t x_size, size_t y_size) {
  /*
   * This convolution implements element-wise convolution and uses exact number of threads. [1 & 2]
   * Below assumptions are considered in this convolution implementation.
   *  - Assumption 1 : GPU cannot expect the size and traits of the filter.
   *  - Assumption 2 : The only known fact is that the shape of the filter is square.
   *  - Assumption 3 : GPU cannot expect the size and traits of the image.
   */
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;
  uint32_t x_bud = blockDim.x + 2 * filter->bud_size;
  extern __block__ uint8_t dave[]; // shape : (blockDim.x + filter->size - 1, filter->size)

  // Fill the dave
  size_t dave_x_size = blockDim.x + filter->size - 1;
  uint32_t ptr;
  uint8_t val;
  for (int dave_y_pos = 0; dave_y_pos < filter->size; dave_y_pos += 1) {
    int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + dave_y_pos;
    if (src_y_pos < 0) {
      ptr = x_pos;
    } else if (src_y_pos >= y_size) {
      ptr = (y_size - 1) * x_size + x_pos;
    } else {
      ptr = src_y_pos * x_size + x_pos;
    }
    dave[dave_y_pos * dave_x_size + threadIdx.x + filter->bud_size] = in[ptr];
  }
  if (threadIdx.x < filter->bud_size) {
    if (blockIdx.x != 0) {
      for (int dave_y_pos = 0; dave_y_pos < filter->size; dave_y_pos += 1) {
        int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + dave_y_pos;
        if (src_y_pos < 0) {
          val = in[(size_t)blockDim.x * blockIdx.x - filter->bud_size + threadIdx.x];
        } else if (src_y_pos >= y_size) {
          val = in[x_size * (y_size - 1) + (size_t)blockDim.x * blockIdx.x - filter->bud_size + threadIdx.x];
        } else {
          val = in[src_y_pos * x_size + (size_t)blockDim.x * blockIdx.x - filter->bud_size
            + threadIdx.x];
        }
        dave[dave_y_pos * dave_x_size + threadIdx.x] = val;
      }
    } else {
      for (int dave_y_pos = 0; dave_y_pos < filter->size; dave_y_pos += 1) {
        int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + dave_y_pos;
        if (src_y_pos < 0) {
          ptr = threadIdx.x;
        } else if (src_y_pos >= y_size) {
          ptr = (y_size - 1) * x_size + threadIdx.x;
        } else {
          ptr = src_y_pos * x_size + threadIdx.x;
        }
        dave[dave_y_pos * dave_x_size + threadIdx.x] = in[ptr];
      }
    }
    if ((gridDim.x - 1) > blockIdx.x) {
      for (int dave_y_pos = 0; dave_y_pos < filter->size; dave_y_pos += 1) {
        int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + dave_y_pos;
        if (src_y_pos < 0) {
          val = in[((size_t)blockIdx.x + 1) * blockDim.x + threadIdx.x];
        } else if (src_y_pos >= y_size) {
          val = in[(y_size - 1) * x_size + ((size_t)blockIdx.x + 1) * blockDim.x + threadIdx.x];
        } else {
          val = in[src_y_pos * x_size + ((size_t)blockIdx.x + 1) * blockDim.x + threadIdx.x];
        }
        dave[((size_t)dave_y_pos + 1) * dave_x_size - filter->bud_size + threadIdx.x] = val;
      }
    } else {
      for (int dave_y_pos = 0; dave_y_pos < filter->size; dave_y_pos += 1) {
        int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + dave_y_pos;
        if (src_y_pos < 0) {
          ptr = x_size - filter->bud_size + threadIdx.x;
        } else if (src_y_pos >= y_size) {
          ptr = x_size * y_size - filter->bud_size + threadIdx.x;
        } else {
          ptr = (src_y_pos + 1) * x_size - filter->bud_size + threadIdx.x;
        }
        dave[((size_t)dave_y_pos + 1) * dave_x_size - filter->bud_size + threadIdx.x] = in[ptr];
      }
    }
  }
  __syncthreads();

  // Calculate convolution
  int32_t sum = 0;
  for (size_t fy = 0; fy < filter->size; fy += 1) {
    for (size_t fx = 0; fx < filter->size; fx += 1) {
      sum += filter->kernel[fy * filter->size + fx] * dave[threadIdx.x + fy * x_bud + fx];
    }
  }
  sum /= filter->kernel_sum;
  *(out + index) = sum >= 0 ? (sum <= 255 ? sum : 255) : 0;
};
void cuda::conv_be_unsigned __make_host_f__(conv_be_unsigned, uint8_t, int8_t, uint8_t);

__global__ void conv_bb_unsigned(uint8_t* in, const filt::Filter<int8_t>* filter, uint8_t* out,
                                 size_t x_size, size_t y_size) {
  /*
   * This convolution implements element-wise convolution using bigger threads block. [3]
   * Below assumptions are considered in this convolution implementation.
   *  - Assumption 1 : GPU cannot expect the size and traits of the filter.
   *  - Assumption 2 : The only known fact is that the shape of the filter is square.
   *  - Assumption 3 : GPU cannot expect the size and traits of the image.
   */
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;
  uint32_t x_bud = blockDim.x + 2 * filter->bud_size;
  int dave_y_pos = threadIdx.z;
  extern __block__ uint8_t dave[]; // shape : (blockDim.x + filter->size - 1, filter->size)

  // Fill the dave
  size_t dave_x_size = blockDim.x + filter->size - 1;
  uint32_t ptr;
  uint8_t val;
  int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + dave_y_pos;
  if (src_y_pos < 0) {
    ptr = x_pos;
  } else if (src_y_pos >= y_size) {
    ptr = (y_size - 1) * x_size + x_pos;
  } else {
    ptr = src_y_pos * x_size + x_pos;
  }
  dave[dave_y_pos * dave_x_size + threadIdx.x + filter->bud_size] = in[ptr];
  if (threadIdx.x < filter->bud_size) {
    if (blockIdx.x != 0) {
      int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + dave_y_pos;
      if (src_y_pos < 0) {
        val = in[(size_t)blockDim.x * blockIdx.x - filter->bud_size + threadIdx.x];
      } else if (src_y_pos >= y_size) {
        val = in[x_size * (y_size - 1) + (size_t)blockDim.x * blockIdx.x - filter->bud_size + threadIdx.x];
      } else {
        val = in[src_y_pos * x_size + (size_t)blockDim.x * blockIdx.x - filter->bud_size
          + threadIdx.x];
      }
      dave[dave_y_pos * dave_x_size + threadIdx.x] = val;
    } else {
      int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + dave_y_pos;
      if (src_y_pos < 0) {
        ptr = threadIdx.x;
      } else if (src_y_pos >= y_size) {
        ptr = (y_size - 1) * x_size + threadIdx.x;
      } else {
        ptr = src_y_pos * x_size + threadIdx.x;
      }
      dave[dave_y_pos * dave_x_size + threadIdx.x] = in[ptr];
    }
    if ((gridDim.x - 1) > blockIdx.x) {
      int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + dave_y_pos;
      if (src_y_pos < 0) {
        val = in[((size_t)blockIdx.x + 1) * blockDim.x + threadIdx.x];
      } else if (src_y_pos >= y_size) {
        val = in[(y_size - 1) * x_size + ((size_t)blockIdx.x + 1) * blockDim.x + threadIdx.x];
      } else {
        val = in[src_y_pos * x_size + ((size_t)blockIdx.x + 1) * blockDim.x + threadIdx.x];
      }
      dave[((size_t)dave_y_pos + 1) * dave_x_size - filter->bud_size + threadIdx.x] = val;
    } else {
      int src_y_pos = (int)blockIdx.y - (int)filter->bud_size + dave_y_pos;
      if (src_y_pos < 0) {
        ptr = x_size - filter->bud_size + threadIdx.x;
      } else if (src_y_pos >= y_size) {
        ptr = x_size * y_size - filter->bud_size + threadIdx.x;
      } else {
        ptr = (src_y_pos + 1) * x_size - filter->bud_size + threadIdx.x;
      }
      dave[((size_t)dave_y_pos + 1) * dave_x_size - filter->bud_size + threadIdx.x] = in[ptr];
    }
  }
  __syncthreads();

  // Calculate convolution
  int32_t sum = 0;
  for (size_t fy = 0; fy < filter->size; fy += 1) {
    for (size_t fx = 0; fx < filter->size; fx += 1) {
      sum += filter->kernel[fy * filter->size + fx] * dave[threadIdx.x + fy * x_bud + fx];
    }
  }
  sum /= filter->kernel_sum;
  *(out + index) = sum >= 0 ? (sum <= 255 ? sum : 255) : 0;
};
void cuda::conv_bb_unsigned __make_host_f__(conv_bb_unsigned, uint8_t, int8_t, uint8_t);

void cuda::conv_be_signed __cuda_todo_f__(conv_be_signed, uint8_t, int8_t, int8_t);

//////////////////
// Sobel Filter //
//////////////////

void cuda::sobel_zp __cuda_todo__(sobel_zp, uint8_t, uint8_t);
void cuda::sobel_be __cuda_todo__(sobel_be, uint8_t, uint8_t);

///////////////////
// Median Filter //
///////////////////

void cuda::median_3tap __cuda_todo__(median_3tap, uint8_t, uint8_t);
void cuda::median_5tap __cuda_todo__(median_5tap, uint8_t, uint8_t);

__global__ void median_5tap_horz(uint8_t* in, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;

  if (x_pos < x_size - 4) {
    uint8_t point1 = in[index];
    uint8_t point2 = in[index + 1];
    uint8_t point3 = in[index + 2];
    uint8_t point4 = in[index + 3];
    uint8_t point5 = in[index + 4];
    cudaSort2(point1, point2);
    cudaSort2(point4, point5);
    cudaSort2(point3, point5);
    cudaSort2(point3, point4);
    cudaSort2(point1, point4);
    cudaSort2(point1, point3);
    cudaSort2(point2, point5);
    cudaSort2(point2, point4);
    cudaSort2(point2, point3);
    out[index] = point3;
  }
};
void cuda::median_5tap_horz __make_host__(median_5tap_horz, uint8_t, uint8_t);

void cuda::median_3by3 __cuda_todo__(median_3by3, uint8_t, uint8_t);

////////////////////
// Multiplication //
////////////////////

__global__ void multiply(uint8_t* in1, uint8_t* in2, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;

  if (x_pos < x_size) {
    for (size_t i = 0; i < x_size; i += 1) {
      out[index] += in1[y_pos * x_size + i] * in2[i * x_size + x_pos];
    }
  }
};
void cuda::multiply __make_host_2__(multiply, uint8_t, uint8_t);

/////////////
// Scaling //
/////////////

void cuda::scale_05 __cuda_todo__(scale_05, uint8_t, uint8_t);
void cuda::scale_13 __cuda_todo__(scale_13, uint8_t, uint8_t);
void cuda::scale_24 __cuda_todo__(scale_24, uint8_t, uint8_t);
