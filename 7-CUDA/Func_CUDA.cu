#include "hip/hip_runtime.h"
#include "Core.h"
#include "Functions.cuh"
#include "Util.cuh"

/********************
 ** CUDA Functions **
 ********************/

///////////////
// Inversion //
///////////////

__global__ void invert_8b(uint8_t* in, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = y_pos * x_size + x_pos;
  
  if (x_pos < x_size) {
    out[index] = 255 - in[index];
  }
};
void cuda::invert_8b __make_host__(invert_8b, uint8_t, uint8_t);

//////////////////
// Accumulation //
//////////////////

void cuda::accumulation_16b __cuda_todo__(accumulation_16b, uint16_t, uint64_t);

//////////////
// Flipping //
//////////////

__global__ void xflip(uint8_t* in, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;

  if (x_pos < x_size) {
    out[y_pos * x_size + x_size - x_pos - 1] = in[x_size * y_pos + x_pos];
  }
};
void cuda::xflip __make_host__(xflip, uint8_t, uint8_t);

__global__ void yflip(uint8_t* in, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;

  if (x_pos < x_size) {
    out[(y_size - y_pos - 1) * x_size + x_pos] = in[x_size * y_pos + x_pos];
  }
};
void cuda::yflip __make_host__(yflip, uint8_t, uint8_t);

//////////////
// Addition //
//////////////

__global__ void add_8b(uint8_t* in1, uint8_t* in2, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;

  if (x_pos < x_size) {
    out[index] = in1[index] / 2 + in2[index] / 2;
  }
};
void cuda::add_8b __make_host_2__(add_8b, uint8_t, uint8_t);

__global__ void add_16b(uint8_t* in1, uint8_t* in2, uint16_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;

  if (x_pos < x_size) {
    out[index] = (uint16_t)in1[index] + (uint16_t)in2[index];
  }
};
void cuda::add_16b __make_host_2__(add_16b, uint8_t, uint16_t);

/////////////////
// Dot Product //
/////////////////

__global__ void dot(uint8_t* in1, uint8_t* in2, uint64_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;
  /*__shared__ uint64_t good_boy[512];
  if (index < 512) good_boy[index] = 0;
  __syncthreads();

  if (x_pos < x_size) {
    // printf("%llx %llx\n", (uint64_t)in1[index], (uint64_t)in1[index]);
    atomicAdd(&good_boy[index % 512], (uint64_t)in1[index] * (uint64_t)in2[index]);
  }
  __syncthreads();
  if (index < 512) {
    printf("%llx ", good_boy[index]);
    atomicAdd(out, good_boy[index]);
  }*/
  if (x_pos < x_size) {
    atomicAdd(out, (uint64_t)in1[index] * (uint64_t)in2[index]);
  }
};
void cuda::dot __make_host_2__(dot, uint8_t, uint64_t);

///////////////
// Transpose //
///////////////

__global__ void transpose8(uint8_t* in, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;

  if (x_pos < x_size) {
    out[x_pos * x_size + y_pos] = in[x_size * y_pos + x_pos];
  }
};
void cuda::transpose_line_by_line __make_host__(transpose8, uint8_t, uint8_t);
void cuda::transpose_block_8 __make_host__(transpose8, uint8_t, uint8_t);
void cuda::transpose_block_16 __make_host__(transpose8, uint8_t, uint8_t);
void cuda::transpose_block_32 __make_host__(transpose8, uint8_t, uint8_t);
void cuda::transpose_block_64 __make_host__(transpose8, uint8_t, uint8_t);
void cuda::transpose_block_128 __make_host__(transpose8, uint8_t, uint8_t);

__global__ void transpose16(uint16_t* in, uint16_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;

  if (x_pos < x_size) {
    out[x_pos * x_size + y_pos] = in[x_size * y_pos + x_pos];
  }
};
void cuda::transpose16_block_8 __make_host__(transpose16, uint16_t, uint16_t);

__global__ void transpose32(uint32_t* in, uint32_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;

  if (x_pos < x_size) {
    out[x_pos * x_size + y_pos] = in[x_size * y_pos + x_pos];
  }
};
void cuda::transpose32_block_4 __make_host__(transpose32, uint32_t, uint32_t);

/////////////////
// Convolution //
/////////////////

void cuda::conv_zp_unsigned __cuda_todo_f__(conv_zp_unsigned, uint8_t, int8_t, uint8_t);
void cuda::conv_zp_signed __cuda_todo_f__(conv_zp_signed, uint8_t, int8_t, int8_t);
void cuda::conv_be_unsigned __cuda_todo_f__(conv_be_unsigned, uint8_t, int8_t, uint8_t);
void cuda::conv_be_signed __cuda_todo_f__(conv_be_signed, uint8_t, int8_t, int8_t);

//////////////////
// Sobel Filter //
//////////////////

void cuda::sobel_zp __cuda_todo__(sobel_zp, uint8_t, uint8_t);
void cuda::sobel_be __cuda_todo__(sobel_be, uint8_t, uint8_t);

///////////////////
// Median Filter //
///////////////////

void cuda::median_3tap __cuda_todo__(median_3tap, uint8_t, uint8_t);
void cuda::median_5tap __cuda_todo__(median_5tap, uint8_t, uint8_t);
void cuda::median_3by3 __cuda_todo__(median_3by3, uint8_t, uint8_t);

////////////////////
// Multiplication //
////////////////////

__global__ void multiply(uint8_t* in1, uint8_t* in2, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = x_pos + y_pos * x_size;

  out[index] = 0;
  if (x_pos < x_size) {
    for (size_t i = 0; i < x_size; i += 1) {
      out[index] += in1[y_pos * x_size + i] * in2[i * x_size + x_pos];
    }
  }
};
void cuda::multiply __make_host_2__(multiply, uint8_t, uint8_t);

/////////////
// Scaling //
/////////////

void cuda::scale_05 __cuda_todo__(scale_05, uint8_t, uint8_t);
void cuda::scale_13 __cuda_todo__(scale_13, uint8_t, uint8_t);
void cuda::scale_24 __cuda_todo__(scale_24, uint8_t, uint8_t);
