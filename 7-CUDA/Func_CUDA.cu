#include "hip/hip_runtime.h"
#include "Core.h"
#include "Functions.cuh"
#include "Util.cuh"

/********************
 ** CUDA Functions **
 ********************/

///////////////
// Inversion //
///////////////

__global__ void invert_8b(uint8_t* in, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t x_pos = __x_pos__;
  uint32_t y_pos = __y_pos__;
  uint32_t index = __index__;
  
  if (index < x_size * y_size) {
    out[index] = 255 - in[index];
  }
};
void cuda::invert_8b __make_host__(invert_8b, uint8_t, uint8_t);

//////////////////
// Accumulation //
//////////////////

void cuda::accumulation_16b __cuda_todo__(accumulation_16b, uint16_t, uint64_t);

//////////////
// Flipping //
//////////////

void cuda::xflip __cuda_todo__(xflip, uint8_t, uint8_t);
void cuda::yflip __cuda_todo__(xflip, uint8_t, uint8_t);

//////////////
// Addition //
//////////////

void cuda::add_8b __cuda_todo_2__(add_8b, uint8_t, uint8_t);
void cuda::add_16b __cuda_todo_2__(add_16b, uint8_t, uint16_t);

///////////////
// Transpose //
///////////////

void cuda::transpose_line_by_line __cuda_todo__(transpose_line_by_line, uint8_t, uint8_t);
void cuda::transpose_block_8 __cuda_todo__(transpose_block_8, uint8_t, uint8_t);
void cuda::transpose_block_16 __cuda_todo__(transpose_block_16, uint8_t, uint8_t);
void cuda::transpose_block_32 __cuda_todo__(transpose_block_32, uint8_t, uint8_t);
void cuda::transpose_block_64 __cuda_todo__(transpose_block_64, uint8_t, uint8_t);
void cuda::transpose_block_128 __cuda_todo__(transpose_block_128, uint8_t, uint8_t);
void cuda::transpose16_block_8 __cuda_todo__(transpose16_block_8, uint16_t, uint16_t);
void cuda::transpose32_block_4 __cuda_todo__(transpose32_block_4, uint32_t, uint32_t);

/////////////////
// Convolution //
/////////////////

void cuda::conv_zp_unsigned __cuda_todo_f__(conv_zp_unsigned, uint8_t, int8_t, uint8_t);
void cuda::conv_zp_signed __cuda_todo_f__(conv_zp_signed, uint8_t, int8_t, int8_t);
void cuda::conv_be_unsigned __cuda_todo_f__(conv_be_unsigned, uint8_t, int8_t, uint8_t);
void cuda::conv_be_signed __cuda_todo_f__(conv_be_signed, uint8_t, int8_t, int8_t);

//////////////////
// Sobel Filter //
//////////////////

void cuda::sobel_zp __cuda_todo__(sobel_zp, uint8_t, uint8_t);
void cuda::sobel_be __cuda_todo__(sobel_be, uint8_t, uint8_t);

///////////////////
// Median Filter //
///////////////////

void cuda::median_3tap __cuda_todo__(median_3tap, uint8_t, uint8_t);
void cuda::median_5tap __cuda_todo__(median_5tap, uint8_t, uint8_t);
void cuda::median_3by3 __cuda_todo__(median_3by3, uint8_t, uint8_t);

/////////////
// Scaling //
/////////////

void cuda::scale_05 __cuda_todo__(scale_05, uint8_t, uint8_t);
void cuda::scale_13 __cuda_todo__(scale_13, uint8_t, uint8_t);
void cuda::scale_24 __cuda_todo__(scale_24, uint8_t, uint8_t);
