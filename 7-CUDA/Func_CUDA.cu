#include "hip/hip_runtime.h"
#include "Core.h"
#include "Functions.cuh"

__global__ void cuda::invert_8b(uint8_t* in, uint8_t* out, size_t x_size, size_t y_size) {
  uint32_t xWidth = blockDim.x * gridDim.x;
  uint32_t xLoc = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t yLoc = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t index = xLoc + yLoc * xWidth;

  if (index < x_size * y_size) {
    out[index] = 255 - in[index];
  }
};
